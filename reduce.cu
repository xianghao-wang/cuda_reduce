#include "hip/hip_runtime.h"
#include <cstdio>
#include <clocale>
#include <hip/hip_runtime.h>

#include "get_time.h"

#ifndef __OPT__
#define __OPT__ 0
#endif

#define TB_SIZE 1024
#define NITERS 1024

using namespace std;

__global__ void reduce_opt0(double *A, double *blockSums, int n)
{
    unsigned int tid, idx, nThreads;
    unsigned int j, offset;
    __shared__ double cached[TB_SIZE];

    tid = threadIdx.x;
    idx = blockIdx.x * blockDim.x + threadIdx.x;
    nThreads = gridDim.x * blockDim.x;

    // Reduce elements to each threads
    cached[tid] = 0.0;
    j = idx;
    while (j < n)
    {
        cached[tid] += A[j];
        j += nThreads;
    }
    __syncthreads();

    // Reduce threads to a block
    for (offset = 1; offset < blockDim.x; offset *= 2)
    {
        if (tid % (2 * offset) == 0)
        {
            cached[tid] += cached[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
        blockSums[blockIdx.x] = cached[0];
}

__global__ void reduce_opt1(double *A, double *blockSums, int n)
{
    unsigned int tid, idx, nThreads;
    unsigned int j, offset, s;
    __shared__ double cached[TB_SIZE];

    tid = threadIdx.x;
    idx = blockIdx.x * blockDim.x + threadIdx.x;
    nThreads = gridDim.x * blockDim.x;

    // Reduce elements to each threads
    cached[tid] = 0.0;
    j = idx;
    while (j < n)
    {
        cached[tid] += A[j];
        j += nThreads;
    }
    __syncthreads();

    // Reduce threads to a block
    s = blockDim.x / 2;
    for (offset = 1; offset < blockDim.x; offset *= 2)
    {
        if (tid < s)
        {
            cached[(offset * 2) * tid] += cached[(offset * 2) * tid + offset];
        }
        s /= 2;
        __syncthreads();
    }

    if (tid == 0)
        blockSums[blockIdx.x] = cached[0];
}

__global__ void reduce_opt2(double *A, double *blockSums, int n)
{
    unsigned int tid, idx, nThreads;
    unsigned int j, offset;
    __shared__ double cached[TB_SIZE];

    tid = threadIdx.x;
    idx = blockIdx.x * blockDim.x + threadIdx.x;
    nThreads = gridDim.x * blockDim.x;

    // Reduce elements to each threads
    cached[tid] = 0.0;
    j = idx;
    while (j < n)
    {
        cached[tid] += A[j];
        j += nThreads;
    }
    __syncthreads();

    // Reduce threads to a block
    for (offset = blockDim.x / 2; offset > 0; offset /= 2)
    {
        if (tid < offset)
        {
            cached[tid] += cached[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
        blockSums[blockIdx.x] = cached[0];
}


int main(int argc, char **argv)
{
    int n, numBlocks;
    double *A, *blockSums;
    double *A_dev, *blockSums_dev;
    double sum;
    int i;

    timer t;
    double tms;

    hipDeviceReset();
    setlocale(LC_NUMERIC, "");

    // Get dimensions
    n = atoi(argv[1]);
    numBlocks = (n + TB_SIZE -1) / TB_SIZE;

    // Prepare data
    A = new double[n];
    blockSums = new double[numBlocks];
    for (i = 0; i < n; ++ i)
        A[i] = (double) i;
    hipMalloc(&A_dev, n * sizeof(double));
    hipMalloc(&blockSums_dev, numBlocks * sizeof(double));
    hipMemcpy(A_dev, A, n * sizeof(double), hipMemcpyHostToDevice);

    // Print status
    printf("=========================================\n");
    printf("= Running on kernel with optimization %d =\n", __OPT__);
    printf("=========================================\n");
    printf("Total %'d threads are launched\n", numBlocks * TB_SIZE);
    printf("Total %'d blocks are launched with %d block size\n", numBlocks, TB_SIZE);

    // Warmup
    for (i = 0; i < 5; ++ i)
    {
        reduce_opt0<<<numBlocks, TB_SIZE>>>(A_dev, blockSums_dev, n);
        hipDeviceSynchronize();
    }

    // Bench
    t.start();
    for (i = 0; i < NITERS; ++ i) {
#if __OPT__ == 0
        reduce_opt0<<<numBlocks, TB_SIZE>>>(A_dev, blockSums_dev, n);
#elif __OPT__ == 1
        reduce_opt1<<<numBlocks, TB_SIZE>>>(A_dev, blockSums_dev, n);
#elif __OPT__ == 2
        reduce_opt2<<<numBlocks, TB_SIZE>>>(A_dev, blockSums_dev, n);
#else
        printf("Undefined optimization level!\n");
        exit(1);
#endif
        hipDeviceSynchronize();
    }
    tms = t.next_time() * 1e3;


    hipMemcpy(blockSums, blockSums_dev, numBlocks * sizeof(double), hipMemcpyDeviceToHost);

    // Calculate sum
    for (i = 0; i < numBlocks; ++ i)
        sum += blockSums[i];
    printf("Result:     %10e\n", sum);
    printf("Expected:   %10e\n", (n - 1.0) * n / 2.0);
    printf("Average kernel time: %5.3lfms\n", tms / NITERS);

    // Release memory
    hipFree(A_dev);
    hipFree(blockSums_dev);
    delete [] A;
    delete [] blockSums;
}
